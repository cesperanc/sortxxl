#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */
 
 #include <math.h>
 #include <time.h>
 #include <stdio.h>
 #include <stdlib.h>
 #include <cutil_inline.h>
 
 #define 	MAX_THREADS 	128
 #define 	N 		512
 
 int* r_values;
 int* d_values;

 void Init(int* values, int i) {
        srand( time(NULL) );
	printf("\n------------------------------\n");
 
        if (i == 0) {
        // Uniform distribution
                printf("Data set distribution: Uniform\n");
                for (int x = 0; x < N; ++x) {
                        values[x] = rand() % 100;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 1) {
        // Gaussian distribution
        #define MEAN    100
        #define STD_DEV	5 
                printf("Data set distribution: Gaussian\n");
                float r;
                for (int x = 0; x < N; ++x) {
                        r  = (rand()%3 - 1) + (rand()%3 - 1) + (rand()%3 - 1);
                        values[x] = int( round(r * STD_DEV + MEAN) );
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 2) {
        // Bucket distribution
                printf("Data set distribution: Bucket\n");
                int j = 0;
                for (int x = 0; x < N; ++x, ++j) {
                        if (j / 20 < 1)
                                values[x] = rand() % 20;
                        else if (j / 20 < 2)
                                values[x] = rand() % 20 + 20;
                        else if (j / 20 < 3)
                                values[x] = rand() % 20 + 40;
                        else if (j / 20 < 4)
                                values[x] = rand() % 20 + 60;
                        else if (j / 20 < 5)
                                values[x] = rand() % 20 + 80; 
                        if (j == 100)
                                j = 0;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 3) {
        // Sorted distribution
                printf("Data set distribution: Sorted\n");
                /*for (int x = 0; x < N; ++x)
                        print("%d ", values[x]);
		*/
 	}
        else if (i == 4) {
        // Zero distribution
                printf("Data set distribution: Zero\n");
                int r = rand() % 100;
                for (int x = 0; x < N; ++x) {
                        values[x] = r;
                        //printf("%d ", values[x]);
                }
        }
	printf("\n");
}
 
 // Kernel function
 __global__ static void Bitonic_Sort(int* values, int j, int k) {
 	const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

 	if (idx < N) {
 		int ixj = idx^j;
  		if (ixj > idx) {
 			if ((idx&k) == 0 && values[idx] > values[ixj]) {
				//exchange(idx, ixj);
				int tmp = values[idx];
				values[idx] = values[ixj];
				values[ixj] = tmp;
			}
			if ((idx&k) != 0 && values[idx] < values[ixj]) {
				//exchange(idx, ixj);
				int tmp = values[idx];
				values[idx] = values[ixj];
				values[ixj] = tmp;
			}
 		}	
 	}
}
 
 // program main
 int main(int argc, char** argv) {
	printf("./bitonic_sort starting with %d numbers...\n", N);
	srand( time(NULL) );
 	unsigned int hTimer;
 	size_t size = N * sizeof(int);
 	
 	// allocate host memory
 	r_values = (int*)malloc(size);
	
	// allocate device memory
	cutilSafeCall( hipMalloc((void**)&d_values, size) );
 	
	/* Types of data sets to be sorted:
	 *	1. Normal distribution
	 *	2. Gaussian distribution
	 *	3. Bucket distribution
	 *	4. Sorted Distribution
	 *	5. Zero Distribution
	 */

 	for (int i = 0; i < 5; ++i) {
		// initialize data set
 		Init(r_values, i);
 		 
 		// copy data to device
 		cutilSafeCall( hipMemcpy(d_values, r_values, size, hipMemcpyHostToDevice) );

		printf("Beginning kernel execution...\n");

 		cutilCheckError( cutCreateTimer(&hTimer) );
 		cutilSafeCall( hipDeviceSynchronize() );
 		cutilCheckError( cutResetTimer(hTimer) );
 		cutilCheckError( cutStartTimer(hTimer) );

		// execute kernel
		for (int k = 2; k <= N; k <<= 1) {
			for (int j = k >> 1; j > 0; j = j >> 1) {
				if (N < MAX_THREADS)
					Bitonic_Sort <<< 1, N >>> (d_values, j, k);
				else
					Bitonic_Sort <<< N / MAX_THREADS, MAX_THREADS >>> (d_values, j, k);
			}
		}
 		cutilCheckMsg( "Kernel execution failed...\n" );
 
 		cutilSafeCall( hipDeviceSynchronize() );
 		cutilCheckError( cutStopTimer(hTimer) );
 		double gpuTime = cutGetTimerValue(hTimer);
 
 		printf("\nKerned execution completed in %f ms\n", gpuTime);
 
 		// copy data back to host
	 	cutilSafeCall( hipMemcpy(r_values, d_values, size, hipMemcpyDeviceToHost) );

	 	// test print
	 	/*for (int i = 0; i < N; ++i) {
	 		printf("%d ", r_values[i]);
	 	}
	 	printf("\n");
		*/
	
		// test
		printf("\nTesting results...\n");
		for (int x = 0; x < N - 1; x++) {
			if (r_values[x] > r_values[x + 1]) {
				printf("Sorting failed.\n");
				break;
			}
			else
				if (x == N - 2)
					printf("SORTING SUCCESSFUL\n");
		}
	}

 	// free memory
 	cutilSafeCall( hipFree(d_values) );
 	free(r_values);
 	
 	cutilExit(argc, argv);
 	hipDeviceReset();
  
}
